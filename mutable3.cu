#include "hip/hip_runtime.h"
// This file provides two functions to help write GPU elementwise kernels:
//
//   gpu_kernel(TensorIterator iter, <lambda>)
//   gpu_kernel_with_scalars(TensorIterator iter, <lambda>)
//
// The gpu_kernel_with_scalars generates specializations that support a
// single scalar CPU argument, such as from `cuda_tensor + 5`. The CPU scalar
// is lifted to a kernel parameter instead of copying to device memory.
// This should be  used in conjunction with TensorIterator::allow_cpu_scalars_,
// which is the default for TensorIterator::binary_op. Otherwise, all inputs
// and the output must be on the GPU.
//
// For example, to write a reciprocal kernel for GPU float Tensors:
//
//   gpu_kernel(iter, []GPU_LAMBDA(float a) {
//    return 1.0f / a;
//   });
//
// To write a multiplication kernel for GPU float Tensors where one argument
// may be a CPU scalar:
//
//   gpu_kernel_with_scalars(iter, []GPU_LAMBDA(float a, float b) {
//     return a * b;
//   });
//
// See BinaryOpsKernel.cu for the complete implementation
//

#include <type_traits>
#include <tuple>

#include "FunctionTraits.h"

// Marks a lambda as executable on both the host and device. The __host__
// attribute is important so that we can access static type information from
// the host, even if the function is typically only executed on the device.
#ifndef GPU_LAMBDA
#define GPU_LAMBDA __host__ __device__
#endif

#define C10_HOST_DEVICE __host__ __device__
#define C10_RESTRICT __restrict__

static constexpr int launch_size_1d = 512;

using ScalarType = int;

// NOTE: @zasdfgbnm is currently working on rewriting the gpu loops.
// Some of the old codes has been moved to namespace legacy, and
// new codes will be put into namespace modern. These two namespaces
// will coexists for a while until the rewrite is done. Once the rewrite
// is done, we will remove the legacy and modern namespace and everything
// will be in at::native directly.
namespace legacy {

template<int nt, int vt, typename func_t>
__global__ void elementwise_kernel(int N, func_t f) {
  int tid = threadIdx.x;
  int nv = nt * vt;
  int idx = nv * blockIdx.x + tid;
  #pragma unroll
  for (int i = 0; i < vt; i++) {
    if (idx < N) {
      f(idx);
      idx += nt;
    }
  }
}

template<int nt, int vt, typename func_t>
static void launch_kernel(int64_t N, func_t f) {
  dim3 block(nt);
  dim3 grid((N + block.x * vt - 1) / (block.x * vt));
  elementwise_kernel<nt, vt, func_t><<<grid, block>>>(N, f);
}

template <typename traits, typename func_t, typename index_t, size_t... INDEX>
C10_HOST_DEVICE typename traits::result_type
invoke_impl(func_t &f, char *const C10_RESTRICT data[], const index_t strides[], int i,
            std::index_sequence<INDEX...>) {
  return f();
}

template <typename func_t, typename index_t, typename traits = function_traits<func_t>>
C10_HOST_DEVICE typename traits::result_type
invoke(func_t &f, char *const C10_RESTRICT data[], const index_t strides[], int i) {
  using Indices = std::make_index_sequence<traits::arity>;
  return invoke_impl<traits>(f, data, strides, i, Indices{});
}

} // namespace legacy


template <typename func_t>
void gpu_kernel_impl(func_t f) {
  using traits = function_traits<func_t>;
  using arg0_t = typename traits::result_type;
  constexpr int ntensors = traits::arity + 1;

  char *const data[ntensors] = {nullptr};

  int64_t numel = 0;
  if (true) {
    int strides[ntensors];
    for (int i = 0; i < ntensors; i++) {
      strides[i] = 0;
    }

    legacy::launch_kernel<launch_size_1d, 1>(numel, [=]GPU_LAMBDA(int idx) mutable {
        arg0_t* out = (arg0_t*)(data[0] + strides[0] * idx);
        *out = legacy::invoke<func_t>(f, &data.data[1], &strides.data[1], idx);
    });
  }
}

int main() {
    float thread_work_index_ = 0;
    gpu_kernel_impl([thread_work_index_]() mutable -> float {
      return thread_work_index_++;
    });
}
