#include <thread>
#include <stdexcept>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(op)                                  \
  {                                                     \
    auto code = (op);                                   \
    if (code != hipSuccess) {                          \
      throw std::runtime_error(hipGetErrorName(code)); \
    }                                                   \
  }


const int N = 1000;

// auto FLAG = cudaEventDisableTiming | cudaEventInterprocess;
auto FLAG = hipEventDisableTiming;

void code() {
  CHECK_CUDA(hipSetDevice(0));
  hipStream_t stream;
  CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  for (int i = 0; i < N; i++) {
    hipEvent_t event;
    CHECK_CUDA(hipEventCreateWithFlags(&event, FLAG));
    CHECK_CUDA(hipEventRecord(event, stream));
    CHECK_CUDA(hipStreamWaitEvent(stream, event, 0));
    CHECK_CUDA(hipEventDestroy(event));
  }
  CHECK_CUDA(hipStreamDestroy(stream));
}

int main() {
  std::thread thread1(code);
  std::thread thread2(code);
  thread1.join();
  thread2.join();
}
