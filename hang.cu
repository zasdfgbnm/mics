#include <condition_variable>
#include <deque>
#include <mutex>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>

#define CHECK_CUDA(op)                                  \
  {                                                     \
    auto code = (op);                                   \
    if (code != hipSuccess) {                          \
      throw std::runtime_error(hipGetErrorName(code)); \
    }                                                   \
  }

template <typename T>
class Queue {
 public:
  void push(T t) {
    std::unique_lock<std::mutex> lock(mutex_);
    queue_.push_back(std::move(t));
    cv_.notify_all();
  }

  T pop() {
    std::unique_lock<std::mutex> lock(mutex_);
    cv_.wait(lock, [&]() { return !queue_.empty(); });
    T t = std::move(queue_.front());
    queue_.pop_front();
    return t;
  }

 private:
  std::deque<T> queue_;
  std::mutex mutex_;
  std::condition_variable cv_;
};

namespace {
const int dataSize = 256 * 1024; // 256KB
const int numTensors = 1000;
} // namespace

auto FLAG = hipEventDisableTiming | hipEventInterprocess;
// auto FLAG = cudaEventDisableTiming;

void senderCode(
    Queue<hipEvent_t>& senderToReceiver,
    Queue<hipEvent_t>& receiverToSender) {
  CHECK_CUDA(hipSetDevice(0));

  void* ptr;
  hipStream_t stream;

  CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CHECK_CUDA(hipMalloc(&ptr, dataSize));

  for (int i = 0; i < numTensors; i++) {
    hipEvent_t event;
    CHECK_CUDA(hipEventCreateWithFlags(&event, FLAG));
    CHECK_CUDA(hipEventRecord(event, stream));

    senderToReceiver.push(event);
  }

  for (int i = 0; i < numTensors; i++) {
    hipEvent_t event = receiverToSender.pop();

    CHECK_CUDA(hipEventDestroy(event));
  }

  CHECK_CUDA(hipFree(ptr));
  CHECK_CUDA(hipStreamDestroy(stream));
}

void receiverCode(
    Queue<hipEvent_t>& senderToReceiver,
    Queue<hipEvent_t>& receiverToSender) {
  CHECK_CUDA(hipSetDevice(0));
  hipStream_t stream;
  CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  for (int i = 0; i < numTensors; i++) {
    hipEvent_t theirEvent = senderToReceiver.pop();
    CHECK_CUDA(hipStreamWaitEvent(stream, theirEvent, 0));
    receiverToSender.push(theirEvent);
  }
  CHECK_CUDA(hipStreamDestroy(stream));
}

void code3() {
  CHECK_CUDA(hipSetDevice(0));
  for (int i = 0; i < numTensors * 100; i++) {
    hipEvent_t myEvent;
    CHECK_CUDA(hipEventCreateWithFlags(&myEvent, FLAG));
    CHECK_CUDA(hipEventDestroy(myEvent));
  }
}

int main() {
  Queue<hipEvent_t> senderToReceiver;
  Queue<hipEvent_t> receiverToSender;

  std::thread senderThread(
      senderCode, std::ref(senderToReceiver), std::ref(receiverToSender));
  std::thread receiverThread(
      receiverCode, std::ref(senderToReceiver), std::ref(receiverToSender));
  std::thread thread3(code3);

  senderThread.join();
  receiverThread.join();
  thread3.join();
}
