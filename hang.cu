#include <condition_variable>
#include <deque>
#include <mutex>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>

#define CHECK_CUDA(op)                                  \
  {                                                     \
    auto code = (op);                                   \
    if (code != hipSuccess) {                          \
      throw std::runtime_error(hipGetErrorName(code)); \
    }                                                   \
  }

template <typename T>
class Queue {
 public:
  void push(T t) {
    std::unique_lock<std::mutex> lock(mutex_);
    queue_.push_back(std::move(t));
    cv_.notify_all();
  }

  T pop() {
    std::unique_lock<std::mutex> lock(mutex_);
    cv_.wait(lock, [&]() { return !queue_.empty(); });
    T t = std::move(queue_.front());
    queue_.pop_front();
    return t;
  }

 private:
  std::deque<T> queue_;
  std::mutex mutex_;
  std::condition_variable cv_;
};

namespace {
const int dataSize = 256 * 1024; // 256KB
const int numTensors = 1000;
} // namespace

auto FLAG = hipEventDisableTiming | hipEventInterprocess;

void senderCode(
    Queue<hipEvent_t>& senderToReceiver,
    Queue<hipEvent_t>& receiverToSender) {
  CHECK_CUDA(hipSetDevice(0));

  void* ptr;
  hipStream_t stream;

  CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CHECK_CUDA(hipMalloc(&ptr, dataSize));

  for (int i = 0; i < numTensors; i++) {
    hipEvent_t event;
    CHECK_CUDA(hipEventCreateWithFlags(&event, FLAG));
    CHECK_CUDA(hipEventRecord(event, stream));

    senderToReceiver.push(event);
  }

  for (int i = 0; i < numTensors; i++) {
    hipEvent_t event = receiverToSender.pop();

    CHECK_CUDA(hipEventDestroy(event));
  }

  CHECK_CUDA(hipFree(ptr));
  CHECK_CUDA(hipStreamDestroy(stream));
}

void receiverCode(
    Queue<hipEvent_t>& senderToReceiver,
    Queue<hipEvent_t>& receiverToSender) {
  CHECK_CUDA(hipSetDevice(0));

  hipStream_t stream;

  CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  for (int i = 0; i < numTensors; i++) {
    hipEvent_t theirEvent = senderToReceiver.pop();

    CHECK_CUDA(hipStreamWaitEvent(stream, theirEvent, 0));

    hipEvent_t myEvent;
    CHECK_CUDA(hipEventCreateWithFlags(&myEvent, FLAG));
    CHECK_CUDA(hipEventRecord(myEvent, stream));
    CHECK_CUDA(hipEventDestroy(myEvent));

    receiverToSender.push(theirEvent);
  }

  CHECK_CUDA(hipStreamDestroy(stream));
}

int main() {
  Queue<hipEvent_t> senderToReceiver;
  Queue<hipEvent_t> receiverToSender;

  std::thread senderThread(
      senderCode, std::ref(senderToReceiver), std::ref(receiverToSender));
  std::thread receiverThread(
      receiverCode, std::ref(senderToReceiver), std::ref(receiverToSender));

  senderThread.join();
  receiverThread.join();
}
