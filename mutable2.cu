
#include <hip/hip_runtime.h>
template<typename func_t>
__global__ void elementwise_kernel(func_t f) {
  f();
}

template<typename func_t>
void launch_kernel(func_t f) {
  elementwise_kernel<func_t><<<1, 1>>>(f);
}

template<typename func_t>
__host__ __device__ float invoke_impl(func_t &f) {
  return f();
}

template<typename func_t>
__host__ __device__ float invoke(func_t &f) {
  return invoke_impl(f);
}

template<typename func_t>
void gpu_kernel_impl(func_t f) {
  launch_kernel([=]__host__ __device__() mutable {
    float* out = nullptr;
    *out = invoke<func_t>(f);
  });
}

int main() {
  int a = 0;
  gpu_kernel_impl([=]__host__ __device__() mutable { return a++; });
}
