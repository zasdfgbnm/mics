#include <condition_variable>
#include <deque>
#include <mutex>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>

#define CHECK_CUDA(op)                                  \
  {                                                     \
    auto code = (op);                                   \
    if (code != hipSuccess) {                          \
      throw std::runtime_error(hipGetErrorName(code)); \
    }                                                   \
  }

template <typename T>
class Queue {
 public:
  void push(T t) {
    std::unique_lock<std::mutex> lock(mutex_);
    queue_.push_back(std::move(t));
    cv_.notify_all();
  }

  T pop() {
    std::unique_lock<std::mutex> lock(mutex_);
    cv_.wait(lock, [&]() { return !queue_.empty(); });
    T t = std::move(queue_.front());
    queue_.pop_front();
    return t;
  }

 private:
  std::deque<T> queue_;
  std::mutex mutex_;
  std::condition_variable cv_;
};

namespace {
const int dataSize = 256 * 1024; // 256KB
const int numTensors = 1000;
} // namespace

// auto FLAG = cudaEventDisableTiming | cudaEventInterprocess;
auto FLAG = hipEventDisableTiming;

void code1() {
  CHECK_CUDA(hipSetDevice(0));

  void* ptr;
  hipStream_t stream;

  CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CHECK_CUDA(hipMalloc(&ptr, dataSize));

  for (int i = 0; i < numTensors; i++) {
    hipEvent_t event;
    CHECK_CUDA(hipEventCreateWithFlags(&event, FLAG));
    CHECK_CUDA(hipEventRecord(event, stream));
    CHECK_CUDA(hipStreamWaitEvent(stream, event, 0));
    CHECK_CUDA(hipEventDestroy(event));
  }

  CHECK_CUDA(hipFree(ptr));
  CHECK_CUDA(hipStreamDestroy(stream));
}

void code2() {
  CHECK_CUDA(hipSetDevice(0));
  for (int i = 0; i < numTensors * 100; i++) {
    hipEvent_t myEvent;
    CHECK_CUDA(hipEventCreateWithFlags(&myEvent, FLAG));
    CHECK_CUDA(hipEventDestroy(myEvent));
  }
}

int main() {
  Queue<hipEvent_t> senderToReceiver;
  Queue<hipEvent_t> receiverToSender;

  std::thread thread1(code1);
  std::thread thread2(code2);

  thread1.join();
  thread2.join();
}
