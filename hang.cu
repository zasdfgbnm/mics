#include <condition_variable>
#include <deque>
#include <mutex>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>

#define CHECK_CUDA(op)                                  \
  {                                                     \
    auto code = (op);                                   \
    if (code != hipSuccess) {                          \
      throw std::runtime_error(hipGetErrorName(code)); \
    }                                                   \
  }


const int N = 1000;

// auto FLAG = cudaEventDisableTiming | cudaEventInterprocess;
auto FLAG = hipEventDisableTiming;

void code1() {
  CHECK_CUDA(hipSetDevice(0));
  hipStream_t stream;
  CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  for (int i = 0; i < N; i++) {
    hipEvent_t event;
    CHECK_CUDA(hipEventCreateWithFlags(&event, FLAG));
    CHECK_CUDA(hipEventRecord(event, stream));
    CHECK_CUDA(hipStreamWaitEvent(stream, event, 0));
    CHECK_CUDA(hipEventDestroy(event));
  }
  CHECK_CUDA(hipStreamDestroy(stream));
}

void code2() {
  CHECK_CUDA(hipSetDevice(0));
  for (int i = 0; i < N; i++) {
    hipEvent_t myEvent;
    CHECK_CUDA(hipEventCreateWithFlags(&myEvent, FLAG));
    CHECK_CUDA(hipEventDestroy(myEvent));
  }
}

int main() {
  std::thread thread1(code1);
  std::thread thread2(code2);
  thread1.join();
  thread2.join();
}
